
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

__global__ void kernel(int* p)
{
  p[threadIdx.x] = threadIdx.x;
  printf("ahoj\n");  
}

int main(int argc, char** argv)
{
  int* dPtr;
  hipMalloc(&dPtr, sizeof(int) * 10);
  
  kernel<<<dim3(1,1,1), dim3(10,1,1)>>>(dPtr);
  
  int ptr[10];
  hipMemcpy(&ptr[0], dPtr, sizeof(int) * 10, hipMemcpyDeviceToHost);
  
  for (int i = 0; i < 10; i++)
  {
    std::cout << ptr[i] << std::endl;  
  }

  return 0;
}
